
#include <hip/hip_runtime.h>
#include <iostream>

static __global__ void kernel(const float *A, const float *b) {
}

int main(int argc, char** argv) {
    float *d_a, *d_b;
    if(hipMalloc(&d_a, sizeof(float)) != hipSuccess) {
        std::cout << "hipMalloc d_a failed" << std::endl;
	return 1;
    }
    if(hipMalloc(&d_b, sizeof(float)) != hipSuccess) {
        std::cout << "hipMalloc d_b failed" << std::endl;
	hipFree(d_a);
	return 1;
    }
    kernel<<<1, 1>>>(d_a, d_b);
    hipFree(d_a);
    hipFree(d_b);
    std::cout << "done." << std::endl;
}

